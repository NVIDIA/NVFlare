
#include <hip/hip_runtime.h>
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

namespace cgbn {

template<uint32_t denominator>
__device__ __forceinline__ uint32_t static_divide_small(uint32_t numerator) {
  uint32_t est=0xFFFFFFFF/denominator;

  // not exact, but ok for den<2^10 and num<2^20
  return __umulhi((uint32_t)est, numerator+1);
}

template<uint32_t denominator>
__device__ __forceinline__ uint32_t static_remainder_small(uint32_t numerator) {

  // not exact, but ok for den<1024 and num<2^20
  return numerator-static_divide_small<denominator>(numerator)*denominator;
}

}