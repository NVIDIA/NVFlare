#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

namespace cgbn {

template<class core, uint32_t tpi, uint32_t padding>
class dispatch_resolver_t {
  public:
  static const uint32_t BITS=core::BITS;
  static const uint32_t LIMBS=core::LIMBS;
  
  static const uint32_t PAD_THREAD=core::PAD_THREAD;
  static const uint32_t PAD_LIMB=core::PAD_LIMB;
  
  static const uint32_t tpi_ONES=(1ull<<tpi)-1;
  static const uint32_t GROUP_ONES=0xFFFFFFFF / tpi_ONES;

  /****************************************************************
   * returns 1 if all bits are zero
   * returns 0 otherwise
   ****************************************************************/
  __device__ __forceinline__ static int32_t fast_negate(uint32_t &x) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, warp_thread=threadIdx.x & warpSize-1;
    uint32_t lane=1<<warp_thread;
    uint32_t p, c;
    int32_t  result;
    
    p=__ballot_sync(sync, x==0);
    c=(p+GROUP_ONES^p)&lane;
    add_cc(c, 0xFFFFFFFF);
    x=subc_cc(0, x);
    result=__shfl_sync(sync, x, padding, tpi);
    x=(group_thread<padding) ? x : 0;
    return result;
  }
  
  __device__ __forceinline__ static int32_t fast_negate(uint32_t x[LIMBS]) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, group_base=group_thread*LIMBS;
    uint32_t warp_thread=threadIdx.x & warpSize-1, lane=1<<warp_thread;
    uint32_t lor, p, c;
    int32_t  result;
    
    lor=mplor<LIMBS>(x);
    p=__ballot_sync(sync, lor==0);
    c=(p+GROUP_ONES^p)&lane;
    
    chain_t<> chain;
    chain.add(c, 0xFFFFFFFF);
    #pragma unroll
    for(int32_t index=0;index<LIMBS;index++)
      x[index]=chain.sub(0, x[index]);
   
    result=__shfl_sync(sync, x[PAD_LIMB], PAD_THREAD, tpi);
    
    #pragma unroll
    for(int32_t index=0;index<LIMBS;index++)
      x[index]=(group_base<BITS/32-index) ? x[index] : 0;
      
    return result;
  }
        
  /****************************************************************
   * returns 1 if carries out
   * returns 0 otherwise
   ****************************************************************/
  __device__ __forceinline__ static int32_t fast_propagate_add(const uint32_t carry, uint32_t &x) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, warp_thread=threadIdx.x & warpSize-1, lane=1<<warp_thread;
    uint32_t g, p, c;
    int32_t  result;
  
    g=__ballot_sync(sync, carry==1);
    p=__ballot_sync(sync, x==0xFFFFFFFF);
  
    c=lane & (g+g+p ^ p);
    x=x+(c!=0);
  
    result=__shfl_sync(sync, x, padding, tpi);
    x=(group_thread<padding) ? x : 0;
    return result;
  }
  
  __device__ __forceinline__ static int32_t fast_propagate_add(const uint32_t carry, uint32_t x[LIMBS]) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, warp_thread=threadIdx.x & warpSize-1, lane=1<<warp_thread;
    uint32_t land, g, p, c;
    int32_t  result;
    
    land=mpland<LIMBS>(x);
    g=__ballot_sync(sync, carry==1);
    p=__ballot_sync(sync, land==0xFFFFFFFF);
  
    c=lane & (g+g+p ^ p);
    
    x[0]=add_cc(x[0], c!=0);
    #pragma unroll
    for(int32_t index=1;index<LIMBS;index++)
      x[index]=addc_cc(x[index], 0);
  
    result=__shfl_sync(sync, x[PAD_LIMB], PAD_THREAD, tpi);
    x[PAD_LIMB]=(group_thread!=PAD_THREAD) ? x[PAD_LIMB] : 0;
    return result;
  }
  
  /****************************************************************
   * returns 1 if carries out
   * returns 0 otherwise
   ****************************************************************/
  __device__ __forceinline__ static int32_t fast_propagate_sub(const uint32_t carry, uint32_t &x) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1;
    uint32_t lane=1<<(threadIdx.x & warpSize-1);
    uint32_t g, p, c;
    int32_t  result;
    
    g=__ballot_sync(sync, carry==0xFFFFFFFF);
    p=__ballot_sync(sync, x==0);
  
    c=lane & (g+g+p ^ p);
    x=x-(c!=0);
    result=__shfl_sync(sync, x, padding, tpi);
    x=(group_thread<padding) ? x : 0;
    return result & 1;
  }
  
  __device__ __forceinline__ static int32_t fast_propagate_sub(const uint32_t carry, uint32_t x[LIMBS]) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, group_base=group_thread*LIMBS;
    uint32_t warp_thread=threadIdx.x & warpSize-1, lane=1<<warp_thread;
    uint32_t lor, g, p, c;
    int32_t  result;
     
    lor=mplor<LIMBS>(x);
    g=__ballot_sync(sync, carry==0xFFFFFFFF);
    p=__ballot_sync(sync, lor==0);
  
    c=lane & (g+g+p ^ p);
    c=(c==0) ? 0 : 0xFFFFFFFF;
 
    x[0]=add_cc(x[0], c);
    #pragma unroll
    for(int32_t index=1;index<LIMBS;index++)
      x[index]=addc_cc(x[index], c);
  
    result=__shfl_sync(sync, x[PAD_LIMB], PAD_THREAD, tpi);
    #pragma unroll
    for(int32_t index=0;index<LIMBS;index++) 
      x[index]=(group_base<BITS/32-index) ? x[index] : 0;
    return result & 1;
  }

  /****************************************************************
   * returns the high word 
   ****************************************************************/
  __device__ __forceinline__ static int32_t resolve_add(const int32_t carry, uint32_t &x) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, lane=1<<(threadIdx.x & warpSize-1);
    uint32_t g, p, c;
    int32_t  result;
  
    c=__shfl_sync(sync, carry, threadIdx.x-1, tpi);
    c=(group_thread==0) ? 0 : c;
    x=add_cc(x, c);
    c=addc(0, 0);

    g=__ballot_sync(sync, c==1);
    p=__ballot_sync(sync, x==0xFFFFFFFF);
  
    c=lane & (g+g+p ^ p);
    x=x+(c!=0);

    result=__shfl_sync(sync, x, padding, tpi);
    x=(group_thread<padding) ? x : 0;
    return result;
  }

  __device__ __forceinline__ static int32_t resolve_add(const int32_t carry, uint32_t x[LIMBS]) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, lane=1<<(threadIdx.x & warpSize-1);
    uint32_t g, p, c, land;
    int32_t  result;
    
    c=__shfl_sync(sync, carry, threadIdx.x-1, tpi);
    c=(group_thread==0) ? 0 : c;
    x[0]=add_cc(x[0], c);
    #pragma unroll
    for(int32_t index=1;index<LIMBS;index++) 
      x[index]=addc_cc(x[index], 0);
    c=addc(0, 0);
  
    land=mpland<LIMBS>(x);
    g=__ballot_sync(sync, c==1);
    p=__ballot_sync(sync, land==0xFFFFFFFF);
  
    c=lane & (g+g+p ^ p);
    
    x[0]=add_cc(x[0], c!=0);
    #pragma unroll
    for(int32_t index=1;index<LIMBS;index++)
      x[index]=addc_cc(x[index], 0);
    
    result=__shfl_sync(sync, x[PAD_LIMB], PAD_THREAD, tpi);
    x[PAD_LIMB]=(group_thread!=PAD_THREAD) ? x[PAD_LIMB] : 0;
    return result;
  }
  
  /****************************************************************
   * returns the high word 
   ****************************************************************/
  __device__ __forceinline__ static int32_t resolve_sub(const int32_t carry, uint32_t &x) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, lane=1<<(threadIdx.x & warpSize-1);
    uint32_t g, p;
    int32_t  c;
    int32_t  result;
  
    c=__shfl_sync(sync, carry, threadIdx.x-1, tpi);
    c=(group_thread==0) ? 0 : c;
    x=add_cc(x, c);
    c=addc(0, c>>31);

    g=__ballot_sync(sync, c==0xFFFFFFFF);
    p=__ballot_sync(sync, x==0);
  
    c=lane & (g+g+p ^ p);
    x=x-(c!=0);

    result=__shfl_sync(sync, x, padding, tpi);
    x=(group_thread<padding) ? x : 0;
    return result;
  }

  __device__ __forceinline__ static int32_t resolve_sub(const int32_t carry, uint32_t x[LIMBS]) {
    uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & tpi-1, group_base=group_thread*LIMBS;
    uint32_t warp_thread=threadIdx.x & warpSize-1, lane=1<<warp_thread;
    uint32_t g, p, land;
    int32_t  c;
    int32_t  result;
    
    if(group_thread==PAD_THREAD)
      x[PAD_LIMB]=x[PAD_LIMB]+2;

    c=__shfl_sync(sync, carry, threadIdx.x-1, tpi);
    c=(group_thread==0) ? 0 : c;
    x[0]=add_cc(x[0], c);
    c=c>>31;
    #pragma unroll
    for(int32_t index=1;index<LIMBS;index++) 
      x[index]=addc_cc(x[index], c);
    c=addc(0, c);
  
    lor=mplor<limbs>(x);
    g=__ballot_sync(sync, c==0xFFFFFFFF);
    p=__ballot_sync(sync, lor==0);
  
    c=lane & (g+g+p ^ p);
  
    c=(c==0) ? 0 : 0xFFFFFFFF;
    x[0]=add_cc(x[0], c);
    #pragma unroll
    for(int32_t index=1;index<limbs;index++) 
      x[index]=addc_cc(x[index], c);
    
    result=__shfl_sync(sync, x[PAD_LIMB], PAD_THREAD, tpi);
    #pragma unroll
    for(int32_t index=0;index<LIMBS;index++) 
      x[index]=(group_base<BITS/32-index) ? x[index] : 0;
    return result;
  }
  
};

} /* namespace cgbn */

