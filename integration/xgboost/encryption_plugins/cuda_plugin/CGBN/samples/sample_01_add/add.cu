#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/


#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/cpu_support.h"
#include "../utility/cpu_simple_bn_math.h"
#include "../utility/gpu_support.h"

/************************************************************************************************
 *  This example performs component-wise addition of two arrays of 1024-bit bignums.
 *
 *  The example uses a number of utility functions and macros:
 *
 *    random_words(uint32_t *words, uint32_t count)
 *       fills words[0 .. count-1] with random data
 *
 *    add_words(uint32_t *r, uint32_t *a, uint32_t *b, uint32_t count) 
 *       sets bignums r = a+b, where r, a, and b are count words in length
 *
 *    compare_words(uint32_t *a, uint32_t *b, uint32_t count)
 *       compare bignums a and b, where a and b are count words in length.
 *       return 1 if a>b, 0 if a==b, and -1 if b>a
 *    
 *    CUDA_CHECK(call) is a macro that checks a CUDA result for an error,
 *    if an error is present, it prints out the error, call, file and line.
 *
 *    CGBN_CHECK(report) is a macro that checks if a CGBN error has occurred.
 *    if so, it prints out the error, and instance information
 *
 ************************************************************************************************/
 
// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 32
#define BITS 1024
#define INSTANCES 100000

// Declare the instance type
typedef struct {
  cgbn_mem_t<BITS> a;
  cgbn_mem_t<BITS> b;
  cgbn_mem_t<BITS> sum;
} instance_t;

// support routine to generate random instances
instance_t *generate_instances(uint32_t count) {
  instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*count);

  for(int index=0;index<count;index++) {
    random_words(instances[index].a._limbs, BITS/32);
    random_words(instances[index].b._limbs, BITS/32);
  }
  return instances;
}

// support routine to verify the GPU results using the CPU
void verify_results(instance_t *instances, uint32_t count) {
  uint32_t correct[BITS/32];
  
  for(int index=0;index<count;index++) {
    add_words(correct, instances[index].a._limbs, instances[index].b._limbs, BITS/32);
    if(compare_words(correct, instances[index].sum._limbs, BITS/32)!=0) {
      printf("gpu add kernel failed on instance %d\n", index);
      return;
    }
  }
  printf("All results match\n");
}

// helpful typedefs for the kernel
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

// the actual kernel
__global__ void kernel_add(cgbn_error_report_t *report, instance_t *instances, uint32_t count) {
  int32_t instance;
  
  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(instance>=count)
    return;

  context_t      bn_context(cgbn_report_monitor, report, instance);   // construct a context
  env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math
  env_t::cgbn_t  a, b, r;                                             // define a, b, r as 1024-bit bignums

  cgbn_load(bn_env, a, &(instances[instance].a));      // load my instance's a value
  cgbn_load(bn_env, b, &(instances[instance].b));      // load my instance's b value
  cgbn_add(bn_env, r, a, b);                           // r=a+b
  cgbn_store(bn_env, &(instances[instance].sum), r);   // store r into sum
}

int main() {
  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  
  printf("Genereating instances ...\n");
  instances=generate_instances(INSTANCES);
  
  printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*INSTANCES));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*INSTANCES, hipMemcpyHostToDevice));
  
  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  
  printf("Running GPU kernel ...\n");
  // launch with 32 threads per instance, 128 threads (4 instances) per block
  kernel_add<<<(INSTANCES+3)/4, 128>>>(report, gpuInstances, INSTANCES);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
    
  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*INSTANCES, hipMemcpyDeviceToHost));
  
  printf("Verifying the results ...\n");
  verify_results(instances, INSTANCES);
  
  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));
}